#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
__global__ void FPN(int *s, int n)//Find prime numbers
{
	int i = blockIdx.x * blockDim.x + threadIdx.x, j;
	bool t;
	if ((i >= 2) && (i <=n))
	{
		if (i == 2)
		{
			t = true;
		}
		else
		{
			for (j = 2; j < i; j++)
			{
				t = true;
				if (i % j == 0)
				{
					t = false;
					break;
				}
			}
		}
		if (t == true)
		{

			s[i] = i;
		}
		else
		{
			s[i] = 0;
		}
	}
}
__global__ void ttgpu(int *s, int *tt, int n)//Calculate the sum of prime numbers
{
	int i;
	tt[0] = 0;
	for (i = 2; i <= n; i++)
	{
		tt[0] = tt[0] + s[i];
	}
}
int main(void)
{
	//Introduce program
	printf("Program : Find the sum of prime numbers from 1 to n using GPU \n\n");
	//Declare variable
	int n, i, block, * s, * a, * tt;
	clock_t start, end;
	double time_use;
	//Enter the value n (1000 < n < 9999)
	printf_s("Enter n : \nn=");
	scanf_s("%d", &n);
	//Memory allocation
	hipMallocManaged((void**)&s, (n + 1) * sizeof(int));
	hipMallocManaged((void**)&tt, sizeof(int));
	//Start recording time
	start = clock();
	//Find primes and calculate the sum of primes from 1 to n 
	FPN <<<n+1, 1 >>> (s, n);
	ttgpu <<<1, 1 >>> (s, tt, n);
	hipDeviceSynchronize();
	//Finish recording timeinish recording time
	end = clock();
	//Find time GPU used
	time_use = (double)(end - start) / CLOCKS_PER_SEC;
	//Print results
	printf_s("Total number of primes from 1 to %d : %d\n", n, tt[0]);
	printf_s("Time used : %lfs\n",time_use);
	//Release memory
	hipFree(s);
	hipFree(tt);
	return;
}